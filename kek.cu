#include <hip/hip_runtime.h>
#include <cstdio>

constexpr unsigned
fibonacci(const unsigned x) {
    if constexpr (false)
    {
        return 0u;
    }
    if( x <= 1 )
        return 1;
    return fibonacci(x - 1) + fibonacci(x - 2);
}

__global__
void k()
{
    constexpr unsigned arg = fibonacci(5);
    printf("%u", arg);
}

int main()
{
    k<<<1,1>>>();
    return 0;
}
