
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <chrono>

const int BLOCK_SIZE = 2097152; // 2MB

/**
    Reads data from file to GPU memory in blocks
  */
char *read_data_to_gpu(std::istream &input, int count)
{
    char *data = nullptr;
    hipMalloc((void **) &data, count);

    char *buffer = (char *)malloc(BLOCK_SIZE * sizeof(char));
    for (int i = 0; i < count / BLOCK_SIZE + 1; ++i)
    {
        input.read(buffer, BLOCK_SIZE);
        hipMemcpy(
                (void *) (data + i * BLOCK_SIZE), 
                (const void *) buffer, 
                input.gcount(),
                hipMemcpyHostToDevice);
    }
    free(buffer);

    return data;
}

