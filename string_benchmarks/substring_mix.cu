#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <chrono>
#include <fstream>

#include "benchmark_utils.cuh"

#define BLOCKS  16
#define THREADS 64

const int DATA_SIZE = 10485760; // 10MB
const int PATTERN_SIZE = 100; // 100 bytes

enum Benchmark { SPECIALIZED, STANDARD };

__device__ int isSubstring = 0;

__global__ 
__stage(1)
    find_substring_mix(
	    __stage(1) char *string, 
		char *sample, 
		__stage(1) int stringLength, 
		int patternLength)
{
	// Number of symbols of the string processed in this thread
    int packSize = stringLength > BLOCKS*THREADS ? stringLength / (BLOCKS*THREADS) : 1;
	// Index of the beginning of the current block of processed symobols
    int index = (threadIdx.x + blockIdx.x * blockDim.x) * packSize;

    if (index + packSize > BLOCKS*THREADS)
    {
        packSize = BLOCKS*THREADS - index;
    }
	// If no more left to process
    if (packSize == 0)
    {
        return;
    }

    for (int i = index; i < index + packSize; ++i)
    {   
        if (i + patternLength >= stringLength)
        {
            break;
        }

        int j;
        for (j = 0; j < patternLength; ++j)
        {
            if (string[i + j] != sample[j])
            {
                break;
            }
        } 

        if (j == patternLength)
        {
            isSubstring = 1;
        }
    }

    return 0;
}


int main(int argc, char *argv[])
{
    // Reading big data
    ifstream string_file("string.in");
    auto string = read_data_to_gpu(string_file, DATA_SIZE);
    string_file.close();

    // Reading pattern
    ifstream pattern_file("pattern.in");
    auto pattern = read_data_to_gpu(pattern_file, PATTERN_SIZE);
    pattern_file.close();

    auto timerBegin = std::chrono::high_resolution_clock::now();
    switch(argv[1])
    {
        case Benchmark.SPECIALIZED:
            find_substring_mix<<<BLOCKS, THREADS>>>(string, pattern, DATA_SIZE, PATTERN_SIZE);
            break;
        case Benchmark.STANDARD: 
            break;
        default:
            std::cout << "Wrong benchmark type!" << std::endl;
    } 
    hipDeviceSynchronize();
    auto timerEnd = std::chrono::high_resolution_clock::now();

    std::cout << "Execution time: ";
    std::cout << std::chrono::duration_cast<std::chrono::microseconds>(
            timerEnd - timerBegin ).count();
    std::cout << std::endl;

    hipFree(string);
    hipFree(pattern);

    return 0;
}
